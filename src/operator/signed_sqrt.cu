/*!
 * Copyright (c) 2016 by Contributors
 * \file signed_sqrt.cu
 * \brief
 * \author Chen Zhu
*/
#include "./signed_sqrt-inl.h"

namespace mxnet{
namespace op{
template<>
Operator *CreateOp<gpu>(SignedSqrtParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new SignedSqrtOp<gpu, DType>(param);
  })
  return op;
}
} // namespace op
} // namespace mxnet