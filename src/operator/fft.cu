#include "./fft-inl.h"
namespace mxnet {
namespace op{
	template<>
	Operator* CreateOp<gpu>(FFTParam param, int dtype) {
		Operator *op = NULL;
		MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
			op = new FFTOp<gpu, DType>(param);
		})
		return op;
	}
}
}