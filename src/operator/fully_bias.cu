#include "./fully_bias-inl.h"
namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(FullyBiasParam param, int dtype)
{
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new FullyBiasOp<gpu, DType>(param);
  })
  return op;
}
} // namespace op
} // namespace mxnet